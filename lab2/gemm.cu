
#include <hip/hip_runtime.h>
__global__ void gemm(float *__restrict__ feature,
                         float *__restrict__ kernel, float *__restrict__ gemm,
                         int M, int K, int N) {
  float gemm_local[4];
  float gemm_local_rest[4];
  __shared__ float feature_shared[128];
  __shared__ float kernel_shared[128];
  float feature_shared_local[4];
  float kernel_shared_local[4];
  float gemm_local1[4];
  float gemm_local1_rest[4];
  float feature_shared_local1[4];
  float kernel_shared_local1[4];
  if (((int)blockIdx.x) < (M / 16)) {
    for (int i_c_init = 0; i_c_init < 2; ++i_c_init) {
      for (int j_c_init = 0; j_c_init < 2; ++j_c_init) {
        gemm_local[((i_c_init * 2) + j_c_init)] = 0.000000e+00f;
        gemm_local_rest[((i_c_init * 2) + j_c_init)] = 0.000000e+00f;
      }
    }
    for (int rx_outer = 0; rx_outer < ((K + 7) / 8); ++rx_outer) {
      __syncthreads();
      for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
        if ((rx_outer * 8) < (K - ((int)threadIdx.x))) {
          feature_shared[(((((int)threadIdx.y) * 16) + (ax0_inner * 8)) +
                          ((int)threadIdx.x))] =
              feature[(((rx_outer * 8) + ((((((int)blockIdx.x) * 16) +
                                            (((int)threadIdx.y) * 2)) +
                                           ax0_inner) *
                                          K)) +
                       ((int)threadIdx.x))];
        }
      }
      for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
        if ((rx_outer * 8) < (K - ((int)threadIdx.x))) {
          if (((((int)blockIdx.y) * 16) + (((int)threadIdx.y) * 2)) <
              (N - ax1_inner)) {
            kernel_shared[(
                ((((int)threadIdx.x) * 16) + (((int)threadIdx.y) * 2)) +
                ax1_inner)] =
                kernel[((((((int)blockIdx.y) * 16) + (((int)threadIdx.y) * 2)) +
                         (((rx_outer * 8) + ((int)threadIdx.x)) * N)) +
                        ax1_inner)];
          }
        }
      }
      __syncthreads();
      for (int rx_inner_outer = 0; rx_inner_outer < 4; ++rx_inner_outer) {
        for (int ax0 = 0; ax0 < 2; ++ax0) {
          for (int ax1 = 0; ax1 < 2; ++ax1) {
            if (((rx_outer * 8) + (rx_inner_outer * 2)) < (K - ax1)) {
              feature_shared_local[((ax0 * 2) + ax1)] =
                  feature_shared[((((((int)threadIdx.x) * 16) + (ax0 * 8)) +
                                   (rx_inner_outer * 2)) +
                                  ax1)];
            }
          }
        }
        for (int ax01 = 0; ax01 < 2; ++ax01) {
          for (int ax11 = 0; ax11 < 2; ++ax11) {
            if (((rx_outer * 8) + (rx_inner_outer * 2)) < (K - ax01)) {
              if (((((int)blockIdx.y) * 16) + (((int)threadIdx.y) * 2)) <
                  (N - ax11)) {
                kernel_shared_local[((ax01 * 2) + ax11)] =
                    kernel_shared[((((rx_inner_outer * 32) + (ax01 * 16)) +
                                    (((int)threadIdx.y) * 2)) +
                                   ax11)];
              }
            }
          }
        }
        for (int i_c = 0; i_c < 2; ++i_c) {
          for (int j_c = 0; j_c < 2; ++j_c) {
            for (int rx_inner_inner = 0; rx_inner_inner < 2; ++rx_inner_inner) {
              if (((rx_outer * 8) + (rx_inner_outer * 2)) <
                  (K - rx_inner_inner)) {
                if (((((int)blockIdx.y) * 16) + (((int)threadIdx.y) * 2)) <
                    (N - j_c)) {
                  gemm_local[((i_c * 2) + j_c)] += 
                  feature_shared_local[((i_c * 2) + rx_inner_inner)] * 
                  kernel_shared_local[((rx_inner_inner * 2) + j_c)];
                }
              }
            }
          }
        }
      }
    }
    for (int i_inner_inner = 0; i_inner_inner < 2; ++i_inner_inner) {
      for (int j_inner_inner = 0; j_inner_inner < 2; ++j_inner_inner) {
        if (((((int)blockIdx.y) * 16) + (((int)threadIdx.y) * 2)) <
            (N - j_inner_inner)) {
          gemm[((((((int)blockIdx.y) * 16) + (((int)threadIdx.y) * 2)) +
                 ((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) * 2)) +
                   i_inner_inner) *
                  N)) +
                j_inner_inner)] =
              gemm_local[((i_inner_inner * 2) + j_inner_inner)];
        }
      }
    }
  } else {
    for (int i_c_init1 = 0; i_c_init1 < 2; ++i_c_init1) {
      for (int j_c_init1 = 0; j_c_init1 < 2; ++j_c_init1) {
        gemm_local1[((i_c_init1 * 2) + j_c_init1)] = 0.000000e+00f;
      }
    }
    for (int rx_outer1 = 0; rx_outer1 < ((K + 7) / 8); ++rx_outer1) {
      for (int ax0_inner1 = 0; ax0_inner1 < 2; ++ax0_inner1) {
        if (((((int)blockIdx.x) * 16) + (((int)threadIdx.y) * 2)) <
            (M - ax0_inner1)) {
          if ((rx_outer1 * 8) < (K - ((int)threadIdx.x))) {
            feature_shared[(((((int)threadIdx.y) * 16) + (ax0_inner1 * 8)) +
                            ((int)threadIdx.x))] =
                feature[(((rx_outer1 * 8) + ((((((int)blockIdx.x) * 16) +
                                               (((int)threadIdx.y) * 2)) +
                                              ax0_inner1) *
                                             K)) +
                         ((int)threadIdx.x))];
          }
        }
      }
      for (int ax1_inner1 = 0; ax1_inner1 < 2; ++ax1_inner1) {
        if ((rx_outer1 * 8) < (K - ((int)threadIdx.x))) {
          if (((((int)blockIdx.y) * 16) + (((int)threadIdx.y) * 2)) <
              (N - ax1_inner1)) {
            kernel_shared[(
                ((((int)threadIdx.x) * 16) + (((int)threadIdx.y) * 2)) +
                ax1_inner1)] =
                kernel[((((((int)blockIdx.y) * 16) + (((int)threadIdx.y) * 2)) +
                         (((rx_outer1 * 8) + ((int)threadIdx.x)) * N)) +
                        ax1_inner1)];
          }
        }
      }
      for (int rx_inner_outer1 = 0; rx_inner_outer1 < 4; ++rx_inner_outer1) {
        for (int ax02 = 0; ax02 < 2; ++ax02) {
          for (int ax12 = 0; ax12 < 2; ++ax12) {
            if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) * 2)) <
                (M - ax02)) {
              if (((rx_outer1 * 8) + (rx_inner_outer1 * 2)) < (K - ax12)) {
                feature_shared_local1[((ax02 * 2) + ax12)] =
                    feature_shared[((((((int)threadIdx.x) * 16) + (ax02 * 8)) +
                                     (rx_inner_outer1 * 2)) +
                                    ax12)];
              }
            }
          }
        }
        for (int ax03 = 0; ax03 < 2; ++ax03) {
          for (int ax13 = 0; ax13 < 2; ++ax13) {
            if (((rx_outer1 * 8) + (rx_inner_outer1 * 2)) < (K - ax03)) {
              if (((((int)blockIdx.y) * 16) + (((int)threadIdx.y) * 2)) <
                  (N - ax13)) {
                kernel_shared_local1[((ax03 * 2) + ax13)] =
                    kernel_shared[((((rx_inner_outer1 * 32) + (ax03 * 16)) +
                                    (((int)threadIdx.y) * 2)) +
                                   ax13)];
              }
            }
          }
        }
        for (int i_c1 = 0; i_c1 < 2; ++i_c1) {
          for (int j_c1 = 0; j_c1 < 2; ++j_c1) {
            for (int rx_inner_inner1 = 0; rx_inner_inner1 < 2;
                 ++rx_inner_inner1) {
              if (((rx_outer1 * 8) + (rx_inner_outer1 * 2)) <
                  (K - rx_inner_inner1)) {
                if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) * 2)) <
                    (M - i_c1)) {
                  if (((((int)blockIdx.y) * 16) + (((int)threadIdx.y) * 2)) <
                      (N - j_c1)) {
                    gemm_local1[((i_c1 * 2) + j_c1)] += 
                    feature_shared_local1[((i_c1 * 2) + rx_inner_inner1)] *
                    kernel_shared_local1[((rx_inner_inner1 * 2) + j_c1)];
                  }
                }
              }
            }
          }
        }
      }
    }
    for (int i_inner_inner1 = 0; i_inner_inner1 < 2; ++i_inner_inner1) {
      for (int j_inner_inner1 = 0; j_inner_inner1 < 2; ++j_inner_inner1) {
        if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) * 2)) <
            (M - i_inner_inner1)) {
          if (((((int)blockIdx.y) * 16) + (((int)threadIdx.y) * 2)) <
              (N - j_inner_inner1)) {
            gemm[((((((int)blockIdx.y) * 16) + (((int)threadIdx.y) * 2)) +
                   ((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) * 2)) +
                     i_inner_inner1) *
                    N)) +
                  j_inner_inner1)] =
                gemm_local1[((i_inner_inner1 * 2) + j_inner_inner1)];
          }
        }
      }
    }
  }
}
